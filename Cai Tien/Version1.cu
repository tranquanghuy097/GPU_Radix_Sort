#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdint.h>
#include <thrust/device_vector.h>
#include <thrust/copy.h>
#include <thrust/sort.h>

__device__ void partition_by_bit(int *values, int bit);
__device__  int plus_scan(int *x);
__device__ void radix_sort(int *values);


#define CHECK(call)                                                            \
{                                                                              \
    const hipError_t error = call;                                            \
    if (error != hipSuccess)                                                  \
    {                                                                          \
        fprintf(stderr, "Error: %s:%d, ", __FILE__, __LINE__);                 \
        fprintf(stderr, "code: %d, reason: %s\n", error,                       \
                hipGetErrorString(error));                                    \
        exit(1);                                                               \
    }                                                                          \
}

struct GpuTimer
{
    hipEvent_t start;
    hipEvent_t stop;

    GpuTimer()
    {
        hipEventCreate(&start);
        hipEventCreate(&stop);
    }

    ~GpuTimer()
    {
        hipEventDestroy(start);
        hipEventDestroy(stop);
    }

    void Start()
    {
        hipEventRecord(start, 0);
        hipEventSynchronize(start);
    }

    void Stop()
    {
        hipEventRecord(stop, 0);
    }

    float Elapsed()
    {
        float elapsed;
        hipEventSynchronize(stop);
        hipEventElapsedTime(&elapsed, start, stop);
        return elapsed;
    }
};

__device__  int plus_scan(int *x)
{
    int i = threadIdx.x; // id of thread executing this instance
    int n = blockDim.x;  // total number of threads in this block
    int offset;          // distance between elements to be added

    for( offset = 1; offset < n; offset *= 2) {
        int t;

        if ( i >= offset ) 
            t = x[i-offset];
        
        __syncthreads();

        if ( i >= offset ) 
            x[i] = t + x[i]; 

        __syncthreads();
    }
    return x[i];
}

__device__ void partition_by_bit(int *values, int bit)
{
    int thread = threadIdx.x;
    int size = blockDim.x;
    int x_i = values[thread];          
    int p_i = (x_i >> bit) & 1; 

        values[thread] = p_i;  
        __syncthreads();

        int T_before = plus_scan(values);
        int T_total  = values[size-1];

        int F_total  = size - T_total;
        __syncthreads();
        if ( p_i )
        {
            values[T_before-1 + F_total] = x_i;
        }
        else
        {
            values[thread - T_before] = x_i;
        }
    
}

__device__ void radix_sort(int *values)
{
    int  bit;
    for( bit = 0; bit < 32; ++bit )
    {
        partition_by_bit(values, bit);
        __syncthreads();
    }
}

__global__ void sortBlk(int *in, int n, int *sortedBlocks, int bit, int nBins)
{
    extern __shared__ int s[];
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if(i < n)
    {
        s[threadIdx.x] = (in[i] >> bit) & (nBins - 1);
    }
    __syncthreads();
    radix_sort(s);
    __syncthreads();
    if(i < n)
    {
        sortedBlocks[i] =  s[threadIdx.x];
    }
    __syncthreads();
}

__global__ void computeHistKernel(int * in, int n, int * hist, int nBins, int gridSize)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;
	if(i < n)
	{
		atomicAdd(&hist[blockIdx.x + in[i] * gridSize], 1);
	}
}

__global__ void scanBlkKernel(int * in, int n, int * out)
{   
    //TODO
	extern __shared__ int s[];
	int i = blockIdx.x * blockDim.x + threadIdx.x;
	if(i < n)
        s[threadIdx.x] = in[i];
    else
        s[threadIdx.x] = 0;
	__syncthreads();
	int temp;
	for(int stride = 1; stride < blockDim.x; stride *= 2)
	{
		if(threadIdx.x >= stride)
			temp = s[threadIdx.x - stride];
		__syncthreads();
		if(threadIdx.x >= stride)
			s[threadIdx.x] += temp;
		__syncthreads();
    }
    if(i < n - 1)
        out[i + 1] = s[threadIdx.x];
    out[0] = 0;
}

__global__ void scatterKernel(int * in, int n, int *sortedBlocks, int *histScan, int * out, int gridSize)
{
    extern __shared__ int s[];
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if(i < n)
    {
        s[threadIdx.x] = sortedBlocks[i];
    }
    __syncthreads();
    int before = 0;
    for(int j = threadIdx.x - 1; j >= 0; j--)
        if(s[threadIdx.x] == s[j])
            before++;
    __syncthreads();
    int index = blockIdx.x + sortedBlocks[i] * gridSize;
    int rank = histScan[index] + before;
    out[rank] = in[i];
}

__global__ void computeHistKernel2(int * src, int n, int * hist, int nBins, int bit)
{
    // TODO
    // Each block computes its local hist using atomic on SMEM
	extern __shared__ int s[];
	for(int i = threadIdx.x; i < nBins; i += blockDim.x)
		s[i] = 0;
	__syncthreads();
	int i = blockIdx.x * blockDim.x + threadIdx.x;
	if(i < n)
	{
		int bin = (src[i] >> bit) & (nBins -1);
		atomicAdd(&s[bin], 1);
	}
	__syncthreads();
    // Each block adds its local hist to global hist using atomic on GMEM
	for(int i = threadIdx.x; i < nBins; i += blockDim.x)
		atomicAdd(&hist[i], s[i]);
}

// (Partially) Parallel radix sort: implement parallel histogram and parallel scan in counting sort
// Assume: nBits (k in slides) in {1, 2, 4, 8, 16}
// Why "int * blockSizes"? 
// Because we may want different block sizes for diffrent kernels:
//   blockSizes[0] for the histogram kernel
//   blockSizes[1] for the scan kernel
void sortBit(const uint32_t * in, int n, 
        uint32_t * out, 
        int nBits, int * blockSizes, int bit)
{
    // TODO
	int nBins = 1 << nBits; // 2^nBits
    int * hist = (int *)malloc(nBins * sizeof(int));
    int * histScan = (int *)malloc(nBins * sizeof(int));

    // In each counting sort, we sort data in "src" and write result to "dst"
    // Then, we swap these 2 pointers and go to the next counting sort
    // At first, we assign "src = in" and "dest = out"
    // However, the data pointed by "in" is read-only 
    // --> we create a copy of this data and assign "src" to the address of this copy
    uint32_t * src = (uint32_t *)malloc(n * sizeof(uint32_t));
    memcpy(src, in, n * sizeof(uint32_t));
    uint32_t * originalSrc = src; // Use originalSrc to free memory later
    uint32_t * dst = out;
	uint32_t * temp;
	
	dim3 blockSize1(blockSizes[0]);
	dim3 blockSize2(blockSizes[1]);
	
    // Allocate device memories
	int  * d_hist, *d_histScan, * d_in;
    CHECK(hipMalloc(&d_in, n * sizeof(int)));
    CHECK(hipMalloc(&d_hist, nBins * sizeof(int)));
    CHECK(hipMalloc(&d_histScan, nBins * sizeof(int)));
    
	// Call kernel
	dim3 gridSize1((n - 1) / blockSize1.x + 1);
	dim3 gridSize2((n - 1) / blockSize2.x + 1);
	
	size_t smemSize = nBins*sizeof(int);
	size_t sharedMemorySizeByte = blockSize2.x * sizeof(int);
    
    int *d_blkSums;
    CHECK(hipMalloc(&d_blkSums, gridSize2.x * sizeof(int)));
    

    // TODO: Compute "hist" of the current digit
	CHECK(hipMemcpy(d_in, src, n * sizeof(int), hipMemcpyHostToDevice));

	CHECK(hipMemset(d_hist, 0, nBins * sizeof(int)));
		
	computeHistKernel2<<<gridSize1, blockSize1, smemSize>>>(d_in, n, d_hist, nBins, bit);

    // TODO: Scan "hist" (exclusively) and save the result to "histScan"
    scanBlkKernel<<<gridSize2, blockSize2, sharedMemorySizeByte>>>(d_hist, nBins, d_histScan);
    CHECK(hipMemcpy(hist, d_histScan, nBins * sizeof(int), hipMemcpyDeviceToHost));

    // TODO: From "histScan", scatter elements in "src" to correct locations in "dst"
	for(int i = 0; i < n; i++)
	{
		int bin = (src[i] >> bit) & (nBins -1);
		dst[hist[bin]] = src[i];
		hist[bin]++;
	}
    	
    // TODO: Swap "src" and "dst"
	temp = src;
	src = dst;
    dst = temp;

    // TODO: Copy result to "out"
    memcpy(out, src, n * sizeof(uint32_t));
    
    // Free memories
    free(hist);
    free(histScan);
    free(originalSrc);

	// Free device memories
    CHECK(hipFree(d_in));
	CHECK(hipFree(d_hist));
    CHECK(hipFree(d_histScan))
	CHECK(hipFree(d_blkSums));
}

void sortParallel(const uint32_t * in, int n, 
    uint32_t * out, 
    int nBits, int * blockSizes)
{
// TODO
    int nBins = 1 << nBits; // 2^nBits

    uint32_t * src = (uint32_t *)malloc(n * sizeof(uint32_t));
    memcpy(src, in, n * sizeof(uint32_t));
    uint32_t * k = (uint32_t *)malloc(n * sizeof(uint32_t));
    uint32_t * originalSrc = src; // Use originalSrc to free memory later
    uint32_t * dst = out;
    uint32_t * temp;

    dim3 blockSize1(blockSizes[0]);
    dim3 blockSize2(blockSizes[1]);

    // Allocate device memories
    int  * d_hist, *d_histScan, * d_in, *d_sortedBlocks, *d_out, *d_k;
    CHECK(hipMalloc(&d_in, n * sizeof(int)));
    CHECK(hipMalloc(&d_out, n * sizeof(int)));
    CHECK(hipMalloc(&d_sortedBlocks, n * sizeof(int)));
    CHECK(hipMalloc(&d_k, n * sizeof(int)));

    // Call kernel
    dim3 gridSize1((n - 1) / blockSize1.x + 1);
    dim3 gridSize2((n - 1) / blockSize2.x + 1);

    CHECK(hipMalloc(&d_hist, nBins * gridSize1.x * sizeof(int)));
    CHECK(hipMalloc(&d_histScan, nBins * gridSize1.x * sizeof(int)));

    int * hist = (int *)malloc(nBins * gridSize1.x * sizeof(int));
    int * histScan = (int *)malloc(nBins * gridSize1.x * sizeof(int));

    size_t smemSize = blockSize1.x*sizeof(int);

    uint32_t *block = (uint32_t *)malloc(blockSize1.x * sizeof(int));
    uint32_t *block2 = (uint32_t *)malloc(blockSize1.x * sizeof(int));
    int m = 0;
    int mul;
    

    GpuTimer timer; 
    int i = 0;
       
    for (int bit = 0; bit < sizeof(uint32_t) * 8; bit += nBits)
    {
        printf("%d: \n", i);
        timer.Start();
        CHECK(hipMemcpy(d_in, src, n * sizeof(int), hipMemcpyHostToDevice));
        sortBlk<<<gridSize1, blockSize1, smemSize>>>(d_in, n, d_sortedBlocks, bit, nBins);
        for(int j = 0; j < n; j++)
        {
            block[m] = src[j];
            m++;
            if((j + 1) % blockSize1.x == 0)
            {
                m = 0;
                sortBit(block, blockSize1.x, block2, nBits, blockSizes, bit);
                mul = (j + 1) / blockSize1.x;
                for(int l = j + 1 - blockSize1.x; l < mul * blockSize1.x; l++)
                {
                    k[l] = block2[m];
                    m++;
                }
                m = 0;
            }
        }
        CHECK(hipMemcpy(d_k, k, n * sizeof(int), hipMemcpyHostToDevice));
        timer.Stop();
        printf("Sort block: %.3f ms\n", timer.Elapsed());
       
        // TODO: Compute "hist" of the current digit

        timer.Start();
        CHECK(hipMemset(d_hist, 0, nBins * gridSize1.x * sizeof(int)));
        computeHistKernel<<<gridSize1, blockSize1>>>(d_sortedBlocks, n, d_hist, nBins, gridSize1.x);
        CHECK(hipMemcpy(hist, d_hist, nBins * gridSize1.x * sizeof(int), hipMemcpyDeviceToHost));
        timer.Stop();
        printf("Hist: %.3f ms\n", timer.Elapsed());

        //TODO: Scan "hist" (exclusively) and save the result to "histScan"
        timer.Start();
        histScan[0] = 0;
        for (int bin = 1; bin < nBins * gridSize1.x; bin++)
            histScan[bin] = histScan[bin - 1] + hist[bin - 1];
        CHECK(hipMemcpy(d_histScan, histScan, nBins * gridSize1.x * sizeof(int), hipMemcpyHostToDevice));
        timer.Stop();
        printf("Scan: %.3f ms\n", timer.Elapsed());
        
        // TODO: From "histScan", scatter elements in "src" to correct locations in "dst"
        scatterKernel<<<gridSize1, blockSize1, smemSize>>>(d_k, n, d_sortedBlocks, d_histScan, d_out, gridSize1.x);
        CHECK(hipMemcpy(dst, d_out, n * sizeof(int), hipMemcpyDeviceToHost));
        timer.Stop();
        printf("Scatter: %.3f ms\n", timer.Elapsed());
        
        // TODO: Swap "src" and "dst"
        temp = src;
        src = dst;
        dst = temp;
        i++;
    }

    // TODO: Copy result to "out"
    memcpy(out, src, n * sizeof(uint32_t));

    // Free memories
    free(originalSrc);
    free(block);
    free(block2);

    // Free device memories
    CHECK(hipFree(d_in));
    CHECK(hipFree(d_out));
    CHECK(hipFree(d_hist));
    CHECK(hipFree(d_histScan));
    CHECK(hipFree(d_sortedBlocks));
    CHECK(hipFree(d_k));
}



// (Partially) Parallel radix sort: implement parallel histogram and parallel scan in counting sort
// Assume: nBits (k in slides) in {1, 2, 4, 8, 16}
// Why "int * blockSizes"? 
// Because we may want different block sizes for diffrent kernels:
//   blockSizes[0] for the histogram kernel
//   blockSizes[1] for the scan kernel
void sortByDevice(const uint32_t * in, int n, 
        uint32_t * out, 
        int nBits, int * blockSizes)
{
    // TODO
	thrust::device_vector<uint32_t> dv_out(in, in + n);
	thrust::sort(dv_out.begin(), dv_out.end());
	thrust::copy(dv_out.begin(), dv_out.end(), out);
}

// Radix sort
void sort(const uint32_t * in, int n, 
        uint32_t * out, 
        int nBits,
        bool useDevice=false, int * blockSizes=NULL)
{
    GpuTimer timer; 
    timer.Start();

    if (useDevice == false)
    {
    	printf("\nRadix sort Satish parallel\n");
        sortParallel(in, n, out, nBits, blockSizes);
    }
    else // use device
    {
    	printf("\nRadix sort by device\n");
        sortByDevice(in, n, out, nBits, blockSizes);
    }

    timer.Stop();
    printf("Time: %.3f ms\n", timer.Elapsed());
}

void printDeviceInfo()
{
    hipDeviceProp_t devProv;
    CHECK(hipGetDeviceProperties(&devProv, 0));
    printf("**********GPU info**********\n");
    printf("Name: %s\n", devProv.name);
    printf("Compute capability: %d.%d\n", devProv.major, devProv.minor);
    printf("Num SMs: %d\n", devProv.multiProcessorCount);
    printf("Max num threads per SM: %d\n", devProv.maxThreadsPerMultiProcessor); 
    printf("Max num warps per SM: %d\n", devProv.maxThreadsPerMultiProcessor / devProv.warpSize);
    printf("GMEM: %zu byte\n", devProv.totalGlobalMem);
    printf("SMEM per SM: %zu byte\n", devProv.sharedMemPerMultiprocessor);
    printf("SMEM per block: %zu byte\n", devProv.sharedMemPerBlock);
    printf("****************************\n");
}

void checkCorrectness(uint32_t * out, uint32_t * correctOut, int n)
{
    for (int i = 0; i < n; i++)
    {
        if (out[i] != correctOut[i])
        {
            printf("INCORRECT :(\n");
            printf("%d\n", i);
            printf("%d\n", out[i]);
            printf("%d\n", correctOut[i]);
            return;
        }
    }
    printf("CORRECT :)\n");
}

void printArray(uint32_t * a, int n)
{
    for (int i = 0; i < n; i++)
        printf("%i ", a[i]);
    printf("\n");
}

int main(int argc, char ** argv)
{
    // PRINT OUT DEVICE INFO
    printDeviceInfo();

    // SET UP INPUT SIZE
    int n = (1 << 20);
    //n = 16384;
    //n = 10;
    printf("\nInput size: %d\n", n);

    // ALLOCATE MEMORIES
    size_t bytes = n * sizeof(uint32_t);
    uint32_t * in = (uint32_t *)malloc(bytes);
    uint32_t * out = (uint32_t *)malloc(bytes); // Device result
    uint32_t * correctOut = (uint32_t *)malloc(bytes); // Host result

    // SET UP INPUT DATA
    for (int i = 0; i < n; i++)
        in[i] = rand();
    //printArray(in, n);

    // SET UP NBITS
    int nBits = 4; // Default
    if (argc > 1)
        nBits = atoi(argv[1]);
    printf("\nNum bits per digit: %d\n", nBits);

    // DETERMINE BLOCK SIZES
    int blockSizes[2] = {512, 512}; // One for histogram, one for scan
    if (argc == 4)
    {
        blockSizes[0] = atoi(argv[2]);
        blockSizes[1] = atoi(argv[3]);
    }
    printf("\nHist block size: %d, scan block size: %d\n", blockSizes[0], blockSizes[1]);

    sort(in, n, out, nBits, false, blockSizes);
    //printArray(correctOut, n);
    
    // SORT BY DEVICE
    sort(in, n, correctOut, nBits, true, blockSizes);
    checkCorrectness(out, correctOut, n);

    // FREE MEMORIES 
    free(in);
    free(out);
    free(correctOut);
    
    return EXIT_SUCCESS;
}
