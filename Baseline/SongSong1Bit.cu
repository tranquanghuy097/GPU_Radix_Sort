#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdint.h>
#include <thrust/device_vector.h>
#include <thrust/copy.h>
#include <thrust/sort.h>

#define CHECK(call)                                                            \
{                                                                              \
    const hipError_t error = call;                                            \
    if (error != hipSuccess)                                                  \
    {                                                                          \
        fprintf(stderr, "Error: %s:%d, ", __FILE__, __LINE__);                 \
        fprintf(stderr, "code: %d, reason: %s\n", error,                       \
                hipGetErrorString(error));                                    \
        exit(1);                                                               \
    }                                                                          \
}

struct GpuTimer
{
    hipEvent_t start;
    hipEvent_t stop;

    GpuTimer()
    {
        hipEventCreate(&start);
        hipEventCreate(&stop);
    }

    ~GpuTimer()
    {
        hipEventDestroy(start);
        hipEventDestroy(stop);
    }

    void Start()
    {
        hipEventRecord(start, 0);
        hipEventSynchronize(start);
    }

    void Stop()
    {
        hipEventRecord(stop, 0);
    }

    float Elapsed()
    {
        float elapsed;
        hipEventSynchronize(stop);
        hipEventElapsedTime(&elapsed, start, stop);
        return elapsed;
    }
};

__device__  int plus_scan(int *x)
{
    int i = threadIdx.x; // id of thread executing this instance
    int n = blockDim.x;  // total number of threads in this block
    int offset;          // distance between elements to be added

    for( offset = 1; offset < n; offset *= 2) {
        int t;

        if ( i >= offset ) 
            t = x[i-offset];
        
        __syncthreads();

        if ( i >= offset ) 
            x[i] = t + x[i]; 

        __syncthreads();
    }
    return x[i];
}

__device__ void partition_by_bit(int *values, int bit)
{
    int thread = threadIdx.x;
    int size = blockDim.x;
    int x_i = values[thread];          
    int p_i = (x_i >> bit) & 1;   

    values[thread] = p_i;  

    __syncthreads();

    int T_before = plus_scan(values);
    int T_total  = values[size-1];

    int F_total  = size - T_total;
    __syncthreads();
    if ( p_i )
        values[T_before-1 + F_total] = x_i;
    else
        values[thread - T_before] = x_i;
}

__device__ void radix_sort(int *values)
{
    int  bit;
    for( bit = 0; bit < 32; ++bit )
    {
        partition_by_bit(values, bit);
        __syncthreads();
    }
}

__global__ void sortBlk(int *in, int n, int *out)
{
    extern __shared__ int s[];
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if(i < n)
    {
        s[threadIdx.x] = in[i];
    }
    __syncthreads();
    radix_sort(s);
    __syncthreads();
    if(i < n)
    {
        out[i] =  s[threadIdx.x];
    }
    __syncthreads();
}

void sortParallel(const uint32_t * in, int n, 
    uint32_t * out, 
    int nBits, int * blockSizes)
{
    // TODO
    dim3 blockSize1(blockSizes[0]);
    int *d_in, *d_out;
    CHECK(hipMalloc(&d_in, n * sizeof(int)));
    CHECK(hipMalloc(&d_out, n * sizeof(int)));

    size_t smemSize = blockSize1.x*sizeof(int);

    CHECK(hipMemcpy(d_in, in, n * sizeof(int), hipMemcpyHostToDevice));
    sortBlk<<<1, blockSize1, smemSize>>>(d_in, n, d_out);
    CHECK(hipMemcpy(out, d_out, n * sizeof(int), hipMemcpyDeviceToHost));
}



// (Partially) Parallel radix sort: implement parallel histogram and parallel scan in counting sort
// Assume: nBits (k in slides) in {1, 2, 4, 8, 16}
// Why "int * blockSizes"? 
// Because we may want different block sizes for diffrent kernels:
//   blockSizes[0] for the histogram kernel
//   blockSizes[1] for the scan kernel
void sortByDevice(const uint32_t * in, int n, 
        uint32_t * out, 
        int nBits, int * blockSizes)
{
    // TODO
	thrust::device_vector<uint32_t> dv_out(in, in + n);
	thrust::sort(dv_out.begin(), dv_out.end());
	thrust::copy(dv_out.begin(), dv_out.end(), out);
}

// Radix sort
void sort(const uint32_t * in, int n, 
        uint32_t * out, 
        int nBits,
        bool useDevice=false, int * blockSizes=NULL)
{
    GpuTimer timer; 
    timer.Start();

    if (useDevice == false)
    {
    	printf("\nRadix sort parallel 1 bit\n");
        sortParallel(in, n, out, nBits, blockSizes);
    }
    else // use device
    {
    	printf("\nRadix sort by device\n");
        sortByDevice(in, n, out, nBits, blockSizes);
    }

    timer.Stop();
    printf("Time: %.3f ms\n", timer.Elapsed());
}

void printDeviceInfo()
{
    hipDeviceProp_t devProv;
    CHECK(hipGetDeviceProperties(&devProv, 0));
    printf("**********GPU info**********\n");
    printf("Name: %s\n", devProv.name);
    printf("Compute capability: %d.%d\n", devProv.major, devProv.minor);
    printf("Num SMs: %d\n", devProv.multiProcessorCount);
    printf("Max num threads per SM: %d\n", devProv.maxThreadsPerMultiProcessor); 
    printf("Max num warps per SM: %d\n", devProv.maxThreadsPerMultiProcessor / devProv.warpSize);
    printf("GMEM: %zu byte\n", devProv.totalGlobalMem);
    printf("SMEM per SM: %zu byte\n", devProv.sharedMemPerMultiprocessor);
    printf("SMEM per block: %zu byte\n", devProv.sharedMemPerBlock);
    printf("****************************\n");
}

void checkCorrectness(uint32_t * out, uint32_t * correctOut, int n)
{
    for (int i = 0; i < n; i++)
    {
        if (out[i] != correctOut[i])
        {
            printf("INCORRECT :(\n");
            return;
        }
    }
    printf("CORRECT :)\n");
}

void printArray(uint32_t * a, int n)
{
    for (int i = 0; i < n; i++)
        printf("%i ", a[i]);
    printf("\n");
}

int main(int argc, char ** argv)
{
    // PRINT OUT DEVICE INFO
    printDeviceInfo();

    // SET UP INPUT SIZE
    int n = (1 << 10);
    //n = 10;
    printf("\nInput size: %d\n", n);

    // ALLOCATE MEMORIES
    size_t bytes = n * sizeof(uint32_t);
    uint32_t * in = (uint32_t *)malloc(bytes);
    uint32_t * out = (uint32_t *)malloc(bytes); // Device result
    uint32_t * correctOut = (uint32_t *)malloc(bytes); // Host result

    // DETERMINE BLOCK SIZES
    int blockSizes[2] = {1024, 512}; // One for histogram, one for scan
    if (argc == 4)
    {
        blockSizes[0] = atoi(argv[2]);
        blockSizes[1] = atoi(argv[3]);
    }
    printf("\nHist block size: %d, scan block size: %d\n", blockSizes[0], blockSizes[1]);

    // SET UP INPUT DATA
    for (int i = 0; i < n; i++)
        in[i] = rand();
    //printArray(in, n);

    // SET UP NBITS
    int nBits = 4; // Default
    if (argc > 1)
        nBits = atoi(argv[1]);
    printf("\nNum bits per digit: %d\n", nBits);

    // SORT BY HOST
    sort(in, n, correctOut, nBits, false, blockSizes);
    //printArray(correctOut, n);
    
    // SORT BY DEVICE
    sort(in, n, out, nBits, true, blockSizes);
    checkCorrectness(out, correctOut, n);
    

    // FREE MEMORIES 
    free(in);
    free(out);
    free(correctOut);
    
    return EXIT_SUCCESS;
}
