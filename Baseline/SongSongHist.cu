#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdint.h>
#include <thrust/device_vector.h>
#include <thrust/copy.h>
#include <thrust/sort.h>

#define CHECK(call)                                                            \
{                                                                              \
    const hipError_t error = call;                                            \
    if (error != hipSuccess)                                                  \
    {                                                                          \
        fprintf(stderr, "Error: %s:%d, ", __FILE__, __LINE__);                 \
        fprintf(stderr, "code: %d, reason: %s\n", error,                       \
                hipGetErrorString(error));                                    \
        exit(1);                                                               \
    }                                                                          \
}

struct GpuTimer
{
    hipEvent_t start;
    hipEvent_t stop;

    GpuTimer()
    {
        hipEventCreate(&start);
        hipEventCreate(&stop);
    }

    ~GpuTimer()
    {
        hipEventDestroy(start);
        hipEventDestroy(stop);
    }

    void Start()
    {
        hipEventRecord(start, 0);
        hipEventSynchronize(start);
    }

    void Stop()
    {
        hipEventRecord(stop, 0);
    }

    float Elapsed()
    {
        float elapsed;
        hipEventSynchronize(stop);
        hipEventElapsedTime(&elapsed, start, stop);
        return elapsed;
    }
};

__global__ void computeHistKernel2(int * src, int n, int * hist, int nBins, int bit)
{
    // TODO
    // Each block computes its local hist using atomic on SMEM
	extern __shared__ int s[];
	for(int i = threadIdx.x; i < nBins; i += blockDim.x)
		s[i] = 0;
	__syncthreads();
	int i = blockIdx.x * blockDim.x + threadIdx.x;
	if(i < n)
	{
		int bin = (src[i] >> bit) & (nBins -1);
		atomicAdd(&s[bin], 1);
	}
	__syncthreads();
    // Each block adds its local hist to global hist using atomic on GMEM
	for(int i = threadIdx.x; i < nBins; i += blockDim.x)
		atomicAdd(&hist[i], s[i]);
}

// (Partially) Parallel radix sort: implement parallel histogram and parallel scan in counting sort
// Assume: nBits (k in slides) in {1, 2, 4, 8, 16}
// Why "int * blockSizes"? 
// Because we may want different block sizes for diffrent kernels:
//   blockSizes[0] for the histogram kernel
//   blockSizes[1] for the scan kernel
void sortParallel(const uint32_t * in, int n, 
        uint32_t * out, 
        int nBits, int * blockSizes)
{
    // TODO
	int nBins = 1 << nBits; // 2^nBits
    int * hist = (int *)malloc(nBins * sizeof(int));
    int * histScan = (int *)malloc(nBins * sizeof(int));

    // In each counting sort, we sort data in "src" and write result to "dst"
    // Then, we swap these 2 pointers and go to the next counting sort
    // At first, we assign "src = in" and "dest = out"
    // However, the data pointed by "in" is read-only 
    // --> we create a copy of this data and assign "src" to the address of this copy
    uint32_t * src = (uint32_t *)malloc(n * sizeof(uint32_t));
    memcpy(src, in, n * sizeof(uint32_t));
    uint32_t * originalSrc = src; // Use originalSrc to free memory later
    uint32_t * dst = out;
	uint32_t * temp;
	
	dim3 blockSize1(blockSizes[0]);
	dim3 blockSize2(blockSizes[1]);
	
    // Allocate device memories
	int  * d_hist, *d_histScan, * d_in;
    CHECK(hipMalloc(&d_in, n * sizeof(int)));
    CHECK(hipMalloc(&d_hist, nBins * sizeof(int)));
    CHECK(hipMalloc(&d_histScan, nBins * sizeof(int)));
    
	// Call kernel
	dim3 gridSize1((n - 1) / blockSize1.x + 1);
	dim3 gridSize2((n - 1) / blockSize2.x + 1);
	
	size_t smemSize = nBins*sizeof(int);
    
    int *d_blkSums;
    CHECK(hipMalloc(&d_blkSums, gridSize2.x * sizeof(int)));
    

    // Loop from LSD (Least Significant Digit) to MSD (Most Significant Digit)
    // (Each digit consists of nBits bits)
	// In each loop, sort elements according to the current digit 
	// (using STABLE counting sort)
    for (int bit = 0; bit < sizeof(uint32_t) * 8; bit += nBits)
    {
    	// TODO: Compute "hist" of the current digit
		CHECK(hipMemcpy(d_in, src, n * sizeof(int), hipMemcpyHostToDevice));

		CHECK(hipMemset(d_hist, 0, nBins * sizeof(int)));
		
        computeHistKernel2<<<gridSize1, blockSize1, smemSize>>>(d_in, n, d_hist, nBins, bit);
        CHECK(hipMemcpy(hist, d_hist, nBins * sizeof(int), hipMemcpyDeviceToHost));

    	// TODO: Scan "hist" (exclusively) and save the result to "histScan"
        histScan[0] = 0;
        for(int i = 1; i < nBins; i++)
            histScan[i] = histScan[i - 1] + hist[i - 1];

    	// TODO: From "histScan", scatter elements in "src" to correct locations in "dst"
		for(int i = 0; i < n; i++)
		{
			int bin = (src[i] >> bit) & (nBins -1);
			dst[histScan[bin]] = src[i];
			histScan[bin]++;
		}
    	
    	// TODO: Swap "src" and "dst"
		temp = src;
		src = dst;
        dst = temp;
    }

    // TODO: Copy result to "out"
    memcpy(out, src, n * sizeof(uint32_t));
    
    // Free memories
    free(hist);
    free(histScan);
    free(originalSrc);

	// Free device memories
    CHECK(hipFree(d_in));
	CHECK(hipFree(d_hist));
    CHECK(hipFree(d_histScan))
	CHECK(hipFree(d_blkSums));
}


// (Partially) Parallel radix sort: implement parallel histogram and parallel scan in counting sort
// Assume: nBits (k in slides) in {1, 2, 4, 8, 16}
// Why "int * blockSizes"? 
// Because we may want different block sizes for diffrent kernels:
//   blockSizes[0] for the histogram kernel
//   blockSizes[1] for the scan kernel
void sortByDevice(const uint32_t * in, int n, 
        uint32_t * out, 
        int nBits, int * blockSizes)
{
    // TODO
	thrust::device_vector<uint32_t> dv_out(in, in + n);
	thrust::sort(dv_out.begin(), dv_out.end());
	thrust::copy(dv_out.begin(), dv_out.end(), out);
}

// Radix sort
void sort(const uint32_t * in, int n, 
        uint32_t * out, 
        int nBits,
        bool useDevice=false, int * blockSizes=NULL)
{
    GpuTimer timer; 
    timer.Start();

    if (useDevice == false)
    {
    	printf("\nRadix sort parallel scan hist\n");
        sortParallel(in, n, out, nBits, blockSizes);
    }
    else // use device
    {
    	printf("\nRadix sort by device\n");
        sortByDevice(in, n, out, nBits, blockSizes);
    }

    timer.Stop();
    printf("Time: %.3f ms\n", timer.Elapsed());
}

void printDeviceInfo()
{
    hipDeviceProp_t devProv;
    CHECK(hipGetDeviceProperties(&devProv, 0));
    printf("**********GPU info**********\n");
    printf("Name: %s\n", devProv.name);
    printf("Compute capability: %d.%d\n", devProv.major, devProv.minor);
    printf("Num SMs: %d\n", devProv.multiProcessorCount);
    printf("Max num threads per SM: %d\n", devProv.maxThreadsPerMultiProcessor); 
    printf("Max num warps per SM: %d\n", devProv.maxThreadsPerMultiProcessor / devProv.warpSize);
    printf("GMEM: %zu byte\n", devProv.totalGlobalMem);
    printf("SMEM per SM: %zu byte\n", devProv.sharedMemPerMultiprocessor);
    printf("SMEM per block: %zu byte\n", devProv.sharedMemPerBlock);
    printf("****************************\n");
}

void checkCorrectness(uint32_t * out, uint32_t * correctOut, int n)
{
    for (int i = 0; i < n; i++)
    {
        if (out[i] != correctOut[i])
        {
            printf("INCORRECT :(\n");
            return;
        }
    }
    printf("CORRECT :)\n");
}

void printArray(uint32_t * a, int n)
{
    for (int i = 0; i < n; i++)
        printf("%i ", a[i]);
    printf("\n");
}

int main(int argc, char ** argv)
{
    // PRINT OUT DEVICE INFO
    printDeviceInfo();

    // SET UP INPUT SIZE
    int n = (1 << 20);
    //n = 10;
    printf("\nInput size: %d\n", n);

    // ALLOCATE MEMORIES
    size_t bytes = n * sizeof(uint32_t);
    uint32_t * in = (uint32_t *)malloc(bytes);
    uint32_t * out = (uint32_t *)malloc(bytes); // Device result
    uint32_t * correctOut = (uint32_t *)malloc(bytes); // Host result

    // SET UP INPUT DATA
    for (int i = 0; i < n; i++)
        in[i] = rand();
    //printArray(in, n);

    // SET UP NBITS
    int nBits = 4; // Default
    if (argc > 1)
        nBits = atoi(argv[1]);
    printf("\nNum bits per digit: %d\n", nBits);

    // DETERMINE BLOCK SIZES
    int blockSizes[2] = {512, 512}; // One for histogram, one for scan
    if (argc == 4)
    {
        blockSizes[0] = atoi(argv[2]);
        blockSizes[1] = atoi(argv[3]);
    }
    printf("\nHist block size: %d, scan block size: %d\n", blockSizes[0], blockSizes[1]);

    // SORT BY HOST
    sort(in, n, correctOut, nBits, false, blockSizes);
    //printArray(correctOut, n);
    
    // SORT BY DEVICE
    sort(in, n, out, nBits, true, blockSizes);
    checkCorrectness(out, correctOut, n);

    // FREE MEMORIES 
    free(in);
    free(out);
    free(correctOut);
    
    return EXIT_SUCCESS;
}
